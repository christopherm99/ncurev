#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

#define N 4096

extern "C" {
extern const unsigned long long fatbinData[];
}

extern "C" __global__ void saxpy(float a, float *x, float *y) {
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  y[i] = a * x[i] + y[i];
}

int main(void) {
  float *x, *y, *d_x, *d_y;

  x = (float *)malloc(N * sizeof(float));
  y = (float *)malloc(N * sizeof(float));

  hipInit(0);

  hipDevice_t pdev;
  hipDeviceGet(&pdev, 0);

  hipCtx_t pctx;
  hipCtxCreate(&pctx, 0, pdev);

  hipModule_t mod = 0;
  hipModuleLoadData(&mod, fatbinData);
  hipModuleLoadFatBinary(&mod, fatbinData);

  hipFunction_t f = 0;
  hipModuleGetFunction(&f, mod, "saxpy");
  assert(f != 0);

  hipMalloc((hipDeviceptr_t *)&d_x, N * sizeof(float));
  hipMalloc((hipDeviceptr_t *)&d_y, N * sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  cuMemcpy((hipDeviceptr_t)d_x, (hipDeviceptr_t)x, N * sizeof(float));
  cuMemcpy((hipDeviceptr_t)d_y, (hipDeviceptr_t)y, N * sizeof(float));

  float a = 2.0f;
  void *args[] = {&a, &d_x, &d_y};

  hipModuleLaunchKernel(f, N / 256, 1, 1, 256, 1, 1, 0, 0, args, NULL);

  return 0;
}

